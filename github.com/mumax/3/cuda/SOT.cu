#include "hip/hip_runtime.h"
// Original implementation by Mykola Dvornik for mumax2
// Modified for mumax3 by Arne Vansteenkiste, 2013

#include <stdint.h>
#include "float3.h"
#include "constants.h"

extern "C" __global__ void
addspinorbittorque(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                     float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz, float* __restrict__ jz,
                     float* __restrict__ sigx, float* __restrict__ sigy, float* __restrict__ sigz,
                     float* __restrict__ msatLUT, float* __restrict__ alphaLUT, float flt, 
                     float* __restrict__ tSL, float* __restrict__ tFL, 
                     uint8_t* __restrict__ regions, int N) {

	int I =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (I < N) {
		//I=0;

		float3 m = make_float3(mx[I], my[I], mz[I]);
		float  J = jz[I];
		float3 sigma = normalized(make_float3(sigx[I], sigy[I], sigz[I]));

		// read parameters
		uint8_t region       = regions[I];

		float  Ms           = msatLUT[region];
		float  alpha 		= alphaLUT[region];
		float  thetaSL      = tSL[region];
		float  thetaFL      = tFL[region];



		if (J == 0.0f || Ms == 0.0f) {
			return;
		}

		float beta    = (HBAR / QE) * (J / (2*flt*Ms) );
		float gilb     = -1.0f / (1.0f + alpha * alpha);
		if (J < 0.0f) {
			J = -1.0f*J;		}
		float cnst     = gilb * beta;
		float tauS     = cnst * thetaSL;
		float tauF     = cnst * thetaFL;



		float3 mxsig   = cross(m, sigma);
		float3 mxmxsig = cross(m, mxsig);

		tx[I] += tauF * mxsig.x + tauS * mxmxsig.x;
		ty[I] += tauF * mxsig.y + tauS * mxmxsig.y;
		tz[I] += tauF * mxsig.z + tauS * mxmxsig.z;





	}
}

